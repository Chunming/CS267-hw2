#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include "common.h"

//
//  benchmarking program
//
int main( int argc, char **argv )
{    
    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }
    
    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );
    
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );
    set_size( n );
    init_particles( n, particles );
    
    //
    //  simulate a number of time steps
    //
    double simulation_time = read_timer( );
    for( int step = 0; step < NSTEPS; step++ )
    {
        //
        //  compute forces
        //
        for( int i = 0; i < n; i++ )
        {
            particles[i].ax = particles[i].ay = 0;
            for (int j = 0; j < n; j++ )
                apply_force( particles[i], particles[j] );
        }
        
        //
        //  move particles
        //
        for( int i = 0; i < n; i++ ) 
            move( particles[i] );
        
        //
        //  save if necessary
        //
        if( fsave && (step%SAVEFREQ) == 0 )
            save( fsave, n, particles );
    }
    simulation_time = read_timer( ) - simulation_time;
    
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    free( particles );
    if( fsave )
        fclose( fsave );
    
    return 0;
}
