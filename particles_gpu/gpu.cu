#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

#define EMPTY (-1)

int nthreads = 256; 
int numBinsX = 10;
int numBinsY = 10;
int numBins = 100; // Should be perfect square! TODO: change this
int n = 500; 
int maxParticlesPerBin;

particle_t *particles;
particle_t *d_particles = NULL;
int *binParticlesIds = NULL;
int *d_binParticlesIds = NULL;
int *freeLocationPerBin = NULL;
int *d_freeLocationPerBin = NULL;
int *particlesPerBin = NULL;
int *d_particlesPerBin = NULL;



extern double size;
//
//  benchmarking program
//

__device__ void apply_force_gpu(particle_t &particle, particle_t &neighbor)
{
  double dx = neighbor.x - particle.x;
  double dy = neighbor.y - particle.y;
  double r2 = dx * dx + dy * dy;
  if( r2 > cutoff*cutoff )
      return;
  //r2 = fmax( r2, min_r*min_r );
  r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
  double r = sqrt( r2 );

  //
  //  very simple short-range repulsive force
  //
  double coef = ( 1 - cutoff / r ) / r2 / mass;
  particle.ax += coef * dx;
  particle.ay += coef * dy;

}

__global__ void compute_forces_gpu(int n, particle_t *d_particles, int *d_binParticlesIds, int *d_particlesPerBin, int *d_freeLocationPerBin, int numBinsX, int numBinsY, int maxParticlesPerBin)
{
  // Get thread (particle) ID
	int bid = blockIdx.x; // block idx
  	int tid = threadIdx.x ; // thread idx

	if(bid > numBinsX*numBinsY) return;

  if(tid >= d_particlesPerBin[bid]) return;

	int pid = d_binParticlesIds[bid*maxParticlesPerBin + tid];

  d_particles[pid].ax = d_particles[pid].ay = 0;

	// apply force on each particle within the bin
  for(int j = 0 ; j < d_particlesPerBin[bid] ; j++) 
	{
		int jpid = d_binParticlesIds[bid*maxParticlesPerBin + j];
    	apply_force_gpu(d_particles[pid], particles[jpid]);
	}
	
	// apply force on each particle within the bin to the WEST
	int bdx = bid - 1;
	if(bid%numBinsX != 0) // if not on left edge of space
	{
	  for(int j = 0 ; j < d_particlesPerBin[bdx] ; j++) 
		{
			int jpid = d_binParticlesIds[bdx*maxParticlesPerBin + j];
	    	apply_force_gpu(d_particles[pid], particles[jpid]);
		}
	}
	
	// apply force on each particle within the bin to the EAST
	int bdx = bid + 1;
	if((bid+1)%numBinsX != 0) // if not on right edge
	{
	  for(int j = 0 ; j < d_particlesPerBin[bdx] ; j++) 
		{
			int jpid = d_binParticlesIds[bdx*maxParticlesPerBin + j];
	    	apply_force_gpu(d_particles[pid], particles[jpid]);
		}
	}
	
	// apply force on each particle within the bin NORTH
	int bdx = bid - numBinsX;
	if(bid >= numBinsX) // if bid is not in first row
	{
	  for(int j = 0 ; j < d_particlesPerBin[bdx] ; j++) 
		{
			int jpid = d_binParticlesIds[bdx*maxParticlesPerBin + j];
	    	apply_force_gpu(d_particles[pid], particles[jpid]);
		}
	}
	
	// apply force on each particle within the bin SOUTH
	int bdx = bid + numBinsX;
	if(bdx < (numBins - numBinsX))
	{
	  for(int j = 0 ; j < d_particlesPerBin[bdx] ; j++) 
		{
			int jpid = d_binParticlesIds[bdx*maxParticlesPerBin + j];
	    	apply_force_gpu(d_particles[pid], particles[jpid]);
		}
	}
	
	// apply force on each particle within the bin NE
	int bdx = bid + numBinsX;
	if(bid >= numBinsX && (bid+1)%numBinsX != 0)
	{
	  for(int j = 0 ; j < d_particlesPerBin[bdx] ; j++) 
		{
			int jpid = d_binParticlesIds[bdx*maxParticlesPerBin + j];
	    	apply_force_gpu(d_particles[pid], particles[jpid]);
		}
	}
	
	// apply force on each particle within the bin NW
	int bdx = bid + numBinsX;
	if(bid >= numBinsX && bid%numBinsX != 0)
	{
	  for(int j = 0 ; j < d_particlesPerBin[bdx] ; j++) 
		{
			int jpid = d_binParticlesIds[bdx*maxParticlesPerBin + j];
	    	apply_force_gpu(d_particles[pid], particles[jpid]);
		}
	}
	
	// apply force on each particle within the bin SE
	int bdx = bid + numBinsX;
	if(bdx < (numBins - numBinsX) && (bid+1)%numBinsX != 0)
	{
	  for(int j = 0 ; j < d_particlesPerBin[bdx] ; j++) 
		{
			int jpid = d_binParticlesIds[bdx*maxParticlesPerBin + j];
	    	apply_force_gpu(d_particles[pid], particles[jpid]);
		}
	}
	
	// apply force on each particle within the bin SW
	int bdx = bid + numBinsX;
	if(bdx < (numBins - numBinsX) && bid%numBinsX != 0)
	{
	  for(int j = 0 ; j < d_particlesPerBin[bdx] ; j++) 
		{
			int jpid = d_binParticlesIds[bdx*maxParticlesPerBin + j];
	    	apply_force_gpu(d_particles[pid], particles[jpid]);
		}
	}		
}



__global__ void move_gpu (int n, double size, particle_t *d_particles, int *d_binParticlesIds, int *d_particlesPerBin, int *d_freeLocationPerBin, int numBinsX, int numBinsY, int maxParticlesPerBin)
{

  // Get thread (particle) ID within blocks
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	
	if(bid > numBinsX*numBinsY) return;

  	if(tid >= d_particlesPerBin[bid]) return;

	int pid = d_binParticlesIds[bid*maxParticlesPerBin + tid];

  particle_t *p = &d_particles[pid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x  += p->vx * dt;
    p->y  += p->vy * dt;

    //
    //  bounce from walls
    //
    while( p->x < 0 || p->x > size )
    {
        p->x  = p->x < 0 ? -(p->x) : 2*size-p->x;
        p->vx = -(p->vx);
    }
    while( p->y < 0 || p->y > size )
    {
        p->y  = p->y < 0 ? -(p->y) : 2*size-p->y;
        p->vy = -(p->vy);
    }

}

__device__ transferParticle(int pid, int fromBdx, int toBdx)
{
	
}

void doBinning()
{
	numBins = numBinsX * numBinsY;

	double binWidth = sqrt(density * n);
	
	// find maximum no of particles per bin
	double bin_area = (binWidth*binWidth) / numBins; // area of space = size * size 
	maxParticlesPerBin = 3 * (int)( bin_area / (3.14 * (cutoff/2) * (cutoff/2)) ); // radius of particle = cutoff/2
	
	binParticlesIds = (int*) malloc(numBins * sizeof(int) * maxParticlesPerBin);
	if(binParticlesIds == NULL) printf("\ndoBinning(): malloc failed\n");
	
	hipMalloc((void **) &d_binParticlesIds, numBins * sizeof(int) * maxParticlesPerBin);
	if(d_binParticlesIds == NULL) printf("\ndoBinning(): cudamalloc() failed\n");
	
	freeLocationPerBin = (int*) malloc(numBins * sizeof(int));
	if(freeLocationPerBin == NULL) printf("\ndoBinning(): malloc failed\n");
	
	hipMalloc((void **) &d_freeLocationPerBin, numBins * sizeof(int));
	if(d_freeLocationPerBin == NULL) printf("\ndoBinning(): cudamalloc() failed\n");
	
	particlesPerBin = (int*) malloc(numBins * sizeof(int));
	if(particlesPerBin == NULL) printf("\ndoBinning(): malloc failed\n");
	
	hipMalloc((void **) &d_particlesPerBin, numBins * sizeof(int));
	if(d_particlesPerBin == NULL) printf("\ndoBinning(): cudamalloc() failed\n");
	
	for(int i=0; i<maxParticlesPerBin; i++)
	{
		// set all particles ids to EMPTY
		binParticles[i] = EMPTY;
	}

	for(int i=0; i<numBins; i++)
	{
		freeLocationPerBin[i] = 0;
		particlesPerBin[i] = 0;
	}	
	
	for(int ndx=0; ndx<n; ndx++) // for each particle
	{
		int binx = (particles[ndx].x) / binWidth;
		int biny = (particles[ndx].y) / binWidth;
		int bdx = biny*numBinsX + binx;
		
		// add particle ndx to bin bdx
		int loc = freeLocationPerBin[bdx];
		freeLocationPerBin[bdx]++;
		particlesPerBin[bdx]++;
		binParticles[bdx*maxParticlesPerBin + loc] = particles[ndx];
	}
}

int copyToDevice()
{
	hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);
	hipMemcpy(d_binParticlesIds, binParticlesIds, numBins * sizeof(int) * maxParticlesPerBin, hipMemcpyHostToDevice);
	hipMemcpy(d_freeLocationPerBin, freeLocationPerBin, numBins * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_particlesPerBin, particlesPerBin, numBins * sizeof(int), hipMemcpyHostToDevice);
}

void freeBins()
{
	free(binParticlesIds);
	hipFree(d_binParticlesIds);
	
	free(freeLocationPerBin);
	hipFree(d_freeLocationPerBin);
	
	free(particlesPerBin);
	hipFree(d_particlesPerBin);
}


int main( int argc, char **argv )
{    
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize(); 

    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }
    
    n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );
    
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particles = (particle_t*) malloc( n * sizeof(particle_t) );

    // GPU particle data structure
    hipMalloc((void **) &d_particles, n * sizeof(particle_t));

    set_size( n );

    init_particles( n, particles );

    hipDeviceSynchronize();
    double copy_time = read_timer( );

    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    copy_time = read_timer( ) - copy_time;
    
    //
    //  simulate a number of time steps
    //
    hipDeviceSynchronize();
    double simulation_time = read_timer( );

    for( int step = 0; step < NSTEPS; step++ )
    {
        //
        //  compute forces
        //

	int blks = (n + NUM_THREADS - 1) / NUM_THREADS;
	compute_forces_gpu <<< blks, NUM_THREADS >>> (d_particles, n);
        
        //
        //  move particles
        //
	move_gpu <<< blks, NUM_THREADS >>> (d_particles, n, size);
        
        //
        //  save if necessary
        //
        if( fsave && (step%SAVEFREQ) == 0 ) {
	    // Copy the particles back to the CPU
            hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
            save( fsave, n, particles);
			}
    }
    hipDeviceSynchronize();
    simulation_time = read_timer( ) - simulation_time;
    
    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    free( particles );
    hipFree(d_particles);
    if( fsave )
        fclose( fsave );
    
    return 0;
}
