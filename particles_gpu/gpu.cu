#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

#define NUM_THREADS 25


extern double size;
//
//  benchmarking program
//


__device__ void apply_force_gpu(particle_t &particle, particle_t &neighbor)
{
  double dx = neighbor.x - particle.x;
  double dy = neighbor.y - particle.y;
  double r2 = dx * dx + dy * dy;
  if( r2 > cutoff*cutoff )
      return;
  //r2 = fmax( r2, min_r*min_r );
  r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
  double r = sqrt( r2 );

  //
  //  very simple short-range repulsive force
  //
  double coef = ( 1 - cutoff / r ) / r2 / mass;
  
  particle.ax += coef * dx;
  particle.ay += coef * dy;

}

 __device__ inline void myAtomicAdd(double *address, double value)  //See CUDA official forum
{
    unsigned long long oldval, newval, readback;

    oldval = __double_as_longlong(*address);
    newval = __double_as_longlong(__longlong_as_double(oldval) + value);
    while ((readback=atomicCAS((unsigned long long *)address, oldval, newval)) != oldval)
    {
        oldval = readback;
        newval = __double_as_longlong(__longlong_as_double(oldval) + value);
    }
}

__global__ void compute_forces_gpu(particle_t ** bins, int * binParticleNum, double blks_len)
{

  int tId = threadIdx.x;
  int bIdx = blockIdx.x;
  int bIdy = blockIdx.y;
  int blockId = gridDim.x * bIdx + bIdy;
  int blks_num = gridDim.x;



  if (tId >= binParticleNum[blockId]) return;


  // Check all particles in bth subBlock

  for (int j=0; j<binParticleNum[blockId]; ++j) { // The jth particle in bth bin
       apply_force_gpu(*bins[blockId*NUM_THREADS + tId], *bins[blockId*NUM_THREADS + j]);
  }

  // Compute Forces
  double leftBnd, rightBnd, topBnd, botBnd;
  double leftDist, rightDist, topDist, botDist;
  int bLeft, bRight, bBottom, bTop, bTopLeft, bTopRight, bBotLeft, bBotRight;

  //printf("botBnd is %f \n", botBnd);
  leftBnd = bIdx*blks_len;
  rightBnd = (bIdx*blks_len) + blks_len;
  topBnd = bIdy*blks_len;
  botBnd = bIdy*blks_len + blks_len;

  //printf("botDist is %f \n", botDist);
  leftDist = fabs((bins[blockId*NUM_THREADS + tId]->x) - leftBnd);
  rightDist = fabs((bins[blockId*NUM_THREADS + tId]->x) - rightBnd);
  topDist = fabs((bins[blockId*NUM_THREADS + tId]->y) - topBnd);
  botDist = fabs((bins[blockId*NUM_THREADS + tId]->y) - botBnd);

  // Consider 8 different adjacent subBlocks
  if (leftDist<=cutoff) {
      if (bIdx != 0) { // Left subBlock index is valid
	  bLeft = blockId - blks_num;
	  for (int k=0; k<binParticleNum[bLeft]; ++k) { 
               apply_force_gpu(*bins[blockId*NUM_THREADS + tId],*bins[bLeft*NUM_THREADS + k]);
          }
      }
  }

  if (rightDist<=cutoff) {
      if (bIdx != blks_num-1) { 
          bRight = blockId + blks_num; 
	  for (int k=0; k<binParticleNum[bRight]; ++k) { 
	       apply_force_gpu(*bins[blockId*NUM_THREADS + tId],*bins[bRight*NUM_THREADS + k]);
          }
      }
  }

  if (topDist<=cutoff) {
      if (bIdy != 0) { 
          bTop = blockId - 1; 
          for (int k=0; k<binParticleNum[bTop]; ++k) { 
               apply_force_gpu(*bins[blockId*NUM_THREADS + tId],*bins[bTop*NUM_THREADS + k]);
          }
      }
  }

  if (botDist<=cutoff) {
      if (bIdy != blks_num-1) {
          bBottom = blockId + 1;
          for (int k=0; k<binParticleNum[bBottom]; ++k) { 
               apply_force_gpu(*bins[blockId*NUM_THREADS + tId],*bins[bBottom*NUM_THREADS + k]);
          }
      }
  }

  if (topDist<=cutoff && leftDist<=cutoff) { 
      if (bIdy != 0 && bIdx !=0) {
          bTopLeft = blockId-blks_num-1;     
          for (int k=0; k<binParticleNum[bTopLeft]; ++k) { 
               apply_force_gpu(*bins[blockId*NUM_THREADS + tId],*bins[bTopLeft*NUM_THREADS + k]);
          }
      }
  }

  if (botDist<=cutoff && leftDist<=cutoff) { 
      if (bIdy != blks_num-1 && bIdx != 0) { 
          bBotLeft = blockId-blks_num+1;     
          for (int k=0; k<binParticleNum[bBotLeft]; ++k) { 
               apply_force_gpu(*bins[blockId*NUM_THREADS + tId],*bins[bBotLeft*NUM_THREADS + k]);
          }
      }
  }

  if (topDist<=cutoff && rightDist<=cutoff) { 
      if (bIdy != 0 && bIdx != blks_num-1) {
          bTopRight = blockId+blks_num-1;
          for (int k=0; k<binParticleNum[bTopRight]; ++k) { 
               apply_force_gpu(*bins[blockId*NUM_THREADS + tId],*bins[bTopRight*NUM_THREADS + k]);
          }
      }
  }

	    
  if (botDist<=cutoff && rightDist<=cutoff) { 
      if (bIdy!=blks_num-1 && bIdx!=blks_num-1) {
          bBotRight = blockId+blks_num+1;     
          for (int k=0; k<binParticleNum[bBotRight]; ++k) { 
               apply_force_gpu(*bins[blockId*NUM_THREADS + tId],*bins[bBotRight*NUM_THREADS + k]);
          }
      }
  }


}

__global__ void move_gpu (particle_t ** bins, int * part_num, double size)
{

  int tId = threadIdx.x;
  int bIdx = blockIdx.x;
  int bIdy = blockIdx.y;
  int blockId = gridDim.x * bIdx + bIdy;


  if (tId >= part_num[blockId]) return;

  particle_t * p = bins[blockId*NUM_THREADS + tId];

  //
  //  slightly simplified Velocity Verlet integration
  //  conserves energy better than explicit Euler method
  //
  p->vx += p->ax * dt;
  p->vy += p->ay * dt;
  p->x  += p->vx * dt;
  p->y  += p->vy * dt;

  //
  //  bounce from walls
  //
  while( p->x < 0 || p->x > size )
  {
      p->x  = p->x < 0 ? -(p->x) : 2*size-p->x;
      p->vx = -(p->vx);
  }
  while( p->y < 0 || p->y > size )
  {
      p->y  = p->y < 0 ? -(p->y) : 2*size-p->y;
      p->vy = -(p->vy);
  }

}

__global__ void rebin_gpu (particle_t *particles, particle_t **bins, int *part_nums, int n, double blks_size, int blks_num)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (tid >= n) return;

	int x_bin = particles[tid].x/blks_size;
	int y_bin = particles[tid].y/blks_size;
	int blockId = blks_num * x_bin + y_bin;
	int pos = atomicAdd( part_nums + blockId, 1);

	bins[blockId * NUM_THREADS + pos] = particles + tid;
	particles[tid].ax = particles[tid].ay = 0;

}


int main( int argc, char **argv )
{    

    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize(); 


    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }
    
    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", "gpu.txt" );
    
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );

    set_size( n );

    //creates the correct number of blocks given fixed thread size
    int blks_num = (int) ceil(size / sqrt((NUM_THREADS)*(3.14159)*cutoff*cutoff));
    double blks_size =  size / ((double) blks_num);
    dim3 blks (blks_num, blks_num);
 
    int bin_blks = (n + NUM_THREADS - 1) / NUM_THREADS;

    // GPU particle data structure
    particle_t * d_particles;
    particle_t ** d_blocks;
    int * d_blk_part_num;

    // hipMalloc shit
    hipMalloc((void **) &d_particles, n * sizeof(particle_t));
    hipMalloc((void ***) &d_blocks, blks_num * blks_num * NUM_THREADS * sizeof(particle_t*));
    hipMalloc((void **) &d_blk_part_num, blks_num * blks_num * sizeof(int));


    //initialize shit
    hipMemset(d_blk_part_num, 0, blks_num * blks_num * sizeof(int));
    init_particles( n, particles );
    hipDeviceSynchronize();
    double copy_time = read_timer( );

    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    //bin the particles
    rebin_gpu <<< bin_blks, NUM_THREADS >>> (d_particles, d_blocks, d_blk_part_num, n, blks_size, blks_num);

    //calculate time to bin particles
    hipDeviceSynchronize();
    copy_time = read_timer() - copy_time;
    printf( "CPU-GPU copy time = %g seconds\n", copy_time);

    
    //
    //  simulate a number of time steps
    //
    hipDeviceSynchronize();
    double simulation_time = read_timer( );

    

    for( int step = 0; step < NSTEPS; step++ )
    {

        //
        //rebin particles
        //
	hipMemset(d_blk_part_num, 0, blks_num * blks_num * sizeof(int));
        rebin_gpu <<< bin_blks, NUM_THREADS >>> (d_particles, d_blocks, d_blk_part_num, n, blks_size, blks_num);

	//hipDeviceSynchronize();

        //
        //  compute forces
        //
	compute_forces_gpu <<< blks, NUM_THREADS >>> (d_blocks, d_blk_part_num, blks_size);
	//hipDeviceSynchronize();
	//compute_border_forces_gpu <<< blks, NUM_THREADS >>> (d_blocks, d_blk_part_num, blks_size);
	
        
    	//hipDeviceSynchronize();

        //
        //  move particles
        //
	move_gpu <<< blks, NUM_THREADS >>> (d_blocks, d_blk_part_num, size);

    	//hipDeviceSynchronize();
        
        //
        //  save if necessary
        //
        if( fsave && (step%SAVEFREQ) == 0 ) {
	    // Copy the particles back to the CPU
            hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
            save( fsave, n, particles);
	}
    }
    hipDeviceSynchronize();
    simulation_time = read_timer( ) - simulation_time;
    

    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    free( particles );
    hipFree(d_particles);


    hipFree(d_blocks);
    hipFree(d_blk_part_num);


    if( fsave )
        fclose( fsave );
    
    return 0;
}
